#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <sparse/csr.cuh>

#include <raft/cudart_utils.h>
#include <raft/random/rng.cuh>
#include "test_utils.h"

#include <iostream>
#include <limits>
#include <vector>

constexpr int MAX32 = std::numeric_limits<int>::max();
constexpr int64_t MAX64 = std::numeric_limits<int64_t>::max();

namespace MLCommon {
namespace Sparse {

template <typename Index_>
struct CSRMatrix {
  std::vector<Index_> row_ind;
  std::vector<Index_> row_ind_ptr;
};

template <typename Type_f, typename Index_>
struct CSRMatrixVal {
  std::vector<Index_> row_ind;
  std::vector<Index_> row_ind_ptr;
  std::vector<Type_f> values;
};

/**************************** CSR to COO indices ****************************/

template <typename Index_>
struct CSRtoCOOInputs {
  std::vector<Index_> ex_scan;
  std::vector<Index_> verify;
};

template <typename Index_>
class CSRtoCOOTest : public ::testing::TestWithParam<CSRtoCOOInputs<Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<CSRtoCOOInputs<Index_>>::GetParam();

    hipStreamCreate(&stream);
    raft::allocate(ex_scan, params.ex_scan.size());
    raft::allocate(verify, params.verify.size());
    raft::allocate(result, params.verify.size(), true);
  }

  void Run() {
    Index_ n_rows = params.ex_scan.size();
    Index_ nnz = params.verify.size();

    raft::update_device(ex_scan, params.ex_scan.data(), n_rows, stream);
    raft::update_device(verify, params.verify.data(), nnz, stream);

    csr_to_coo<Index_, 32>(ex_scan, n_rows, result, nnz, stream);

    ASSERT_TRUE(raft::devArrMatch<Index_>(verify, result, nnz,
                                          raft::Compare<float>(), stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  CSRtoCOOInputs<Index_> params;
  hipStream_t stream;
  Index_ *ex_scan, *verify, *result;
};

using CSRtoCOOTestI = CSRtoCOOTest<int>;
TEST_P(CSRtoCOOTestI, Result) { Run(); }

using CSRtoCOOTestL = CSRtoCOOTest<int64_t>;
TEST_P(CSRtoCOOTestL, Result) { Run(); }

const std::vector<CSRtoCOOInputs<int>> csrtocoo_inputs_32 = {
  {{0, 0, 2, 2}, {1, 1, 3}},
  {{0, 4, 8, 9}, {0, 0, 0, 0, 1, 1, 1, 1, 2, 3}},
};
const std::vector<CSRtoCOOInputs<int64_t>> csrtocoo_inputs_64 = {
  {{0, 0, 2, 2}, {1, 1, 3}},
  {{0, 4, 8, 9}, {0, 0, 0, 0, 1, 1, 1, 1, 2, 3}},
};

/*********************** CSR row normalize (max, L1) ***********************/

enum NormalizeMethod { MAX, L1 };

template <typename Type_f, typename Index_>
struct CSRRowNormalizeInputs {
  NormalizeMethod method;
  std::vector<Index_> ex_scan;
  std::vector<Type_f> in_vals;
  std::vector<Type_f> verify;
};

template <typename Type_f, typename Index_>
class CSRRowNormalizeTest
  : public ::testing::TestWithParam<CSRRowNormalizeInputs<Type_f, Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<
      CSRRowNormalizeInputs<Type_f, Index_>>::GetParam();
    hipStreamCreate(&stream);

    raft::allocate(in_vals, params.in_vals.size());
    raft::allocate(verify, params.verify.size());
    raft::allocate(ex_scan, params.ex_scan.size());
    raft::allocate(result, params.verify.size(), true);
  }

  void Run() {
    Index_ n_rows = params.ex_scan.size();
    Index_ nnz = params.in_vals.size();

    raft::update_device(ex_scan, params.ex_scan.data(), n_rows, stream);
    raft::update_device(in_vals, params.in_vals.data(), nnz, stream);
    raft::update_device(verify, params.verify.data(), nnz, stream);

    switch (params.method) {
      case MAX:
        csr_row_normalize_max<32, Type_f>(ex_scan, in_vals, nnz, n_rows, result,
                                          stream);
        break;
      case L1:
        csr_row_normalize_l1<32, Type_f>(ex_scan, in_vals, nnz, n_rows, result,
                                         stream);
        break;
    }

    ASSERT_TRUE(
      raft::devArrMatch<Type_f>(verify, result, nnz, raft::Compare<Type_f>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(in_vals));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRRowNormalizeInputs<Type_f, Index_> params;
  hipStream_t stream;
  Index_ *ex_scan;
  Type_f *in_vals, *result, *verify;
};

using CSRRowNormalizeTestF = CSRRowNormalizeTest<float, int>;
TEST_P(CSRRowNormalizeTestF, Result) { Run(); }

using CSRRowNormalizeTestD = CSRRowNormalizeTest<double, int>;
TEST_P(CSRRowNormalizeTestD, Result) { Run(); }

const std::vector<CSRRowNormalizeInputs<float, int>> csrnormalize_inputs_f = {
  {MAX,
   {0, 4, 8, 9},
   {5.0, 1.0, 0.0, 0.0, 10.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {1.0, 0.2, 0.0, 0.0, 1.0, 0.1, 0.0, 0.0, 1, 0.0}},
  {L1,
   {0, 4, 8, 9},
   {1.0, 1.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {0.5, 0.5, 0.0, 0.0, 0.5, 0.5, 0.0, 0.0, 1, 0.0}},
};
const std::vector<CSRRowNormalizeInputs<double, int>> csrnormalize_inputs_d = {
  {MAX,
   {0, 4, 8, 9},
   {5.0, 1.0, 0.0, 0.0, 10.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {1.0, 0.2, 0.0, 0.0, 1.0, 0.1, 0.0, 0.0, 1, 0.0}},
  {L1,
   {0, 4, 8, 9},
   {1.0, 1.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {0.5, 0.5, 0.0, 0.0, 0.5, 0.5, 0.0, 0.0, 1, 0.0}},
};

/********************************* CSR sum *********************************/

template <typename Type_f, typename Index_>
struct CSRSumInputs {
  CSRMatrixVal<Type_f, Index_> matrix_a;
  CSRMatrixVal<Type_f, Index_> matrix_b;
  CSRMatrixVal<Type_f, Index_> matrix_verify;
};

template <typename Type_f, typename Index_>
class CSRSumTest
  : public ::testing::TestWithParam<CSRSumInputs<Type_f, Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<CSRSumInputs<Type_f, Index_>>::GetParam();
    n_rows = params.matrix_a.row_ind.size();
    nnz_a = params.matrix_a.row_ind_ptr.size();
    nnz_b = params.matrix_b.row_ind_ptr.size();
    nnz_result = params.matrix_verify.row_ind_ptr.size();

    hipStreamCreate(&stream);

    raft::allocate(ind_a, n_rows);
    raft::allocate(ind_ptr_a, nnz_a);
    raft::allocate(values_a, nnz_a);

    raft::allocate(ind_b, n_rows);
    raft::allocate(ind_ptr_b, nnz_b);
    raft::allocate(values_b, nnz_b);

    raft::allocate(ind_verify, n_rows);
    raft::allocate(ind_ptr_verify, nnz_result);
    raft::allocate(values_verify, nnz_result);

    raft::allocate(ind_result, n_rows);
    raft::allocate(ind_ptr_result, nnz_result);
    raft::allocate(values_result, nnz_result);
  }

  void Run() {
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);

    raft::update_device(ind_a, params.matrix_a.row_ind.data(), n_rows, stream);
    raft::update_device(ind_ptr_a, params.matrix_a.row_ind_ptr.data(), nnz_a,
                        stream);
    raft::update_device(values_a, params.matrix_a.values.data(), nnz_a, stream);

    raft::update_device(ind_b, params.matrix_b.row_ind.data(), n_rows, stream);
    raft::update_device(ind_ptr_b, params.matrix_b.row_ind_ptr.data(), nnz_b,
                        stream);
    raft::update_device(values_b, params.matrix_b.values.data(), nnz_b, stream);

    raft::update_device(ind_verify, params.matrix_verify.row_ind.data(), n_rows,
                        stream);
    raft::update_device(ind_ptr_verify, params.matrix_verify.row_ind_ptr.data(),
                        nnz_result, stream);
    raft::update_device(values_verify, params.matrix_verify.values.data(),
                        nnz_result, stream);

    Index_ nnz = csr_add_calc_inds<Type_f, 32>(
      ind_a, ind_ptr_a, values_a, nnz_a, ind_b, ind_ptr_b, values_b, nnz_b,
      n_rows, ind_result, alloc, stream);

    ASSERT_TRUE(nnz == nnz_result);
    ASSERT_TRUE(raft::devArrMatch<Index_>(ind_verify, ind_result, n_rows,
                                          raft::Compare<Index_>()));

    csr_add_finalize<Type_f, 32>(ind_a, ind_ptr_a, values_a, nnz_a, ind_b,
                                 ind_ptr_b, values_b, nnz_b, n_rows, ind_result,
                                 ind_ptr_result, values_result, stream);

    ASSERT_TRUE(raft::devArrMatch<Index_>(ind_ptr_verify, ind_ptr_result, nnz,
                                          raft::Compare<Index_>()));
    ASSERT_TRUE(raft::devArrMatch<Type_f>(values_verify, values_result, nnz,
                                          raft::Compare<Type_f>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(ind_a));
    CUDA_CHECK(hipFree(ind_b));
    CUDA_CHECK(hipFree(ind_result));
    CUDA_CHECK(hipFree(ind_ptr_a));
    CUDA_CHECK(hipFree(ind_ptr_b));
    CUDA_CHECK(hipFree(ind_ptr_verify));
    CUDA_CHECK(hipFree(ind_ptr_result));
    CUDA_CHECK(hipFree(values_a));
    CUDA_CHECK(hipFree(values_b));
    CUDA_CHECK(hipFree(values_verify));
    CUDA_CHECK(hipFree(values_result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRSumInputs<Type_f, Index_> params;
  hipStream_t stream;
  Index_ n_rows, nnz_a, nnz_b, nnz_result;
  Index_ *ind_a, *ind_b, *ind_verify, *ind_result, *ind_ptr_a, *ind_ptr_b,
    *ind_ptr_verify, *ind_ptr_result;
  Type_f *values_a, *values_b, *values_verify, *values_result;
};

using CSRSumTestF = CSRSumTest<float, int>;
TEST_P(CSRSumTestF, Result) { Run(); }

using CSRSumTestD = CSRSumTest<double, int>;
TEST_P(CSRSumTestD, Result) { Run(); }

const std::vector<CSRSumInputs<float, int>> csrsum_inputs_f = {
  {{{0, 4, 8, 9},
    {1, 2, 3, 4, 1, 2, 3, 5, 0, 1},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 4, 8, 9},
    {1, 2, 5, 4, 0, 2, 3, 5, 1, 0},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 5, 10, 12},
    {1, 2, 3, 4, 5, 1, 2, 3, 5, 0, 0, 1, 1, 0},
    {2.0, 2.0, 0.5, 1.0, 0.5, 1.0, 2.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}}},
};
const std::vector<CSRSumInputs<double, int>> csrsum_inputs_d = {
  {{{0, 4, 8, 9},
    {1, 2, 3, 4, 1, 2, 3, 5, 0, 1},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 4, 8, 9},
    {1, 2, 5, 4, 0, 2, 3, 5, 1, 0},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 5, 10, 12},
    {1, 2, 3, 4, 5, 1, 2, 3, 5, 0, 0, 1, 1, 0},
    {2.0, 2.0, 0.5, 1.0, 0.5, 1.0, 2.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}}},
};

/******************************** CSR row op ********************************/

template <typename Type_f, typename Index_>
struct CSRRowOpInputs {
  std::vector<Index_> ex_scan;
  std::vector<Type_f> verify;
};

/** Wrapper to call csr_row_op because the enclosing function of a __device__
 *  lambda cannot have private ot protected access within the class. */
template <typename Type_f, typename Index_>
void csr_row_op_wrapper(const Index_ *row_ind, Index_ n_rows, Index_ nnz,
                        Type_f *result, hipStream_t stream) {
  csr_row_op<Index_, 32>(
    row_ind, n_rows, nnz,
    [result] __device__(Index_ row, Index_ start_idx, Index_ stop_idx) {
      for (Index_ i = start_idx; i < stop_idx; i++) result[i] = row;
    },
    stream);
}

template <typename Type_f, typename Index_>
class CSRRowOpTest
  : public ::testing::TestWithParam<CSRRowOpInputs<Type_f, Index_>> {
 protected:
  void SetUp() override {
    params =
      ::testing::TestWithParam<CSRRowOpInputs<Type_f, Index_>>::GetParam();
    hipStreamCreate(&stream);
    n_rows = params.ex_scan.size();
    nnz = params.verify.size();

    raft::allocate(verify, nnz);
    raft::allocate(ex_scan, n_rows);
    raft::allocate(result, nnz, true);
  }

  void Run() {
    raft::update_device(ex_scan, params.ex_scan.data(), n_rows, stream);
    raft::update_device(verify, params.verify.data(), nnz, stream);

    csr_row_op_wrapper<Type_f, Index_>(ex_scan, n_rows, nnz, result, stream);

    ASSERT_TRUE(
      raft::devArrMatch<Type_f>(verify, result, nnz, raft::Compare<Type_f>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRRowOpInputs<Type_f, Index_> params;
  hipStream_t stream;
  Index_ n_rows, nnz;
  Index_ *ex_scan;
  Type_f *result, *verify;
};

using CSRRowOpTestF = CSRRowOpTest<float, int>;
TEST_P(CSRRowOpTestF, Result) { Run(); }

using CSRRowOpTestD = CSRRowOpTest<double, int>;
TEST_P(CSRRowOpTestD, Result) { Run(); }

const std::vector<CSRRowOpInputs<float, int>> csrrowop_inputs_f = {
  {{0, 4, 8, 9}, {0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 2.0, 3.0}},
};
const std::vector<CSRRowOpInputs<double, int>> csrrowop_inputs_d = {
  {{0, 4, 8, 9}, {0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 2.0, 3.0}},
};

/******************************** adj graph ********************************/

template <typename Index_>
struct CSRAdjGraphInputs {
  Index_ n_rows;
  Index_ n_cols;
  std::vector<Index_> row_ind;
  std::vector<uint8_t> adj;  // To avoid vector<bool> optimization
  std::vector<Index_> verify;
};

template <typename Index_>
class CSRAdjGraphTest
  : public ::testing::TestWithParam<CSRAdjGraphInputs<Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<CSRAdjGraphInputs<Index_>>::GetParam();
    hipStreamCreate(&stream);
    nnz = params.verify.size();

    raft::allocate(row_ind, params.n_rows);
    raft::allocate(adj, params.n_rows * params.n_cols);
    raft::allocate(result, nnz, true);
    raft::allocate(verify, nnz);
  }

  void Run() {
    raft::update_device(row_ind, params.row_ind.data(), params.n_rows, stream);
    raft::update_device(adj, reinterpret_cast<bool *>(params.adj.data()),
                        params.n_rows * params.n_cols, stream);
    raft::update_device(verify, params.verify.data(), nnz, stream);

    csr_adj_graph_batched<Index_, 32>(row_ind, params.n_cols, nnz,
                                      params.n_rows, adj, result, stream);

    ASSERT_TRUE(
      raft::devArrMatch<Index_>(verify, result, nnz, raft::Compare<Index_>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(row_ind));
    CUDA_CHECK(hipFree(adj));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRAdjGraphInputs<Index_> params;
  hipStream_t stream;
  Index_ nnz;
  Index_ *row_ind, *result, *verify;
  bool *adj;
};

using CSRAdjGraphTestI = CSRAdjGraphTest<int>;
TEST_P(CSRAdjGraphTestI, Result) { Run(); }

using CSRAdjGraphTestL = CSRAdjGraphTest<int64_t>;
TEST_P(CSRAdjGraphTestL, Result) { Run(); }

const std::vector<CSRAdjGraphInputs<int>> csradjgraph_inputs_i = {
  {3,
   6,
   {0, 3, 6},
   {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
   {0, 1, 2, 0, 1, 2, 0, 1, 2}},
};
const std::vector<CSRAdjGraphInputs<int64_t>> csradjgraph_inputs_l = {
  {3,
   6,
   {0, 3, 6},
   {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
   {0, 1, 2, 0, 1, 2, 0, 1, 2}},
};

/*********************** Weakly connected components ***********************/

template <typename Index_>
struct WeakCCInputs {
  Index_ N;
  std::vector<int8_t> mask;
  std::vector<CSRMatrix<Index_>> batches;
  std::vector<std::vector<Index_>> verify;
};

/** Wrapper to call weakcc because the enclosing function of a __device__
 *  lambda cannot have private ot protected access within the class. */
template <typename Index_>
void weak_cc_wrapper(Index_ *labels, const Index_ *row_ind,
                     const Index_ *row_ind_ptr, Index_ nnz, Index_ N,
                     Index_ startVertexId, Index_ batchSize, WeakCCState *state,
                     hipStream_t stream, bool *mask) {
  weak_cc_batched<Index_>(
    labels, row_ind, row_ind_ptr, nnz, N, startVertexId, batchSize, state,
    stream, [mask, N] __device__(Index_ global_id) {
      return global_id < N ? __ldg((char *)mask + global_id) : 0;
    });
}

template <typename Index_>
class WeakCCTest : public ::testing::TestWithParam<WeakCCInputs<Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<WeakCCInputs<Index_>>::GetParam();

    CUDA_CHECK(hipStreamCreate(&stream));
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);

    Index_ row_ind_size = params.batches[0].row_ind.size();
    Index_ row_ind_ptr_size = params.batches[0].row_ind_ptr.size();
    for (int i = 1; i < params.batches.size(); i++) {
      row_ind_size =
        max(row_ind_size, (Index_)params.batches[i].row_ind.size());
      row_ind_ptr_size =
        max(row_ind_ptr_size, (Index_)params.batches[i].row_ind_ptr.size());
    }

    raft::allocate(row_ind, row_ind_size);
    raft::allocate(row_ind_ptr, row_ind_ptr_size);
    raft::allocate(result, params.N, true);
    raft::allocate(verify, params.N);
    raft::allocate(mask, params.N);
    raft::allocate(m, 1);
  }

  void Run() {
    params = ::testing::TestWithParam<WeakCCInputs<Index_>>::GetParam();
    Index_ N = params.N;

    WeakCCState state(m);

    raft::update_device(mask, reinterpret_cast<bool *>(params.mask.data()), N,
                        stream);

    Index_ start_id = 0;
    for (int i = 0; i < params.batches.size(); i++) {
      Index_ batch_size = params.batches[i].row_ind.size() - 1;
      Index_ row_ind_size = params.batches[i].row_ind.size();
      Index_ row_ind_ptr_size = params.batches[i].row_ind_ptr.size();

      raft::update_device(row_ind, params.batches[i].row_ind.data(),
                          row_ind_size, stream);
      raft::update_device(row_ind_ptr, params.batches[i].row_ind_ptr.data(),
                          row_ind_ptr_size, stream);
      raft::update_device(verify, params.verify[i].data(), N, stream);

      weak_cc_wrapper<Index_>(result, row_ind, row_ind_ptr, row_ind_ptr_size, N,
                              start_id, batch_size, &state, stream, mask);

      hipStreamSynchronize(stream);
      ASSERT_TRUE(
        raft::devArrMatch<Index_>(verify, result, N, raft::Compare<Index_>()));

      start_id += batch_size;
    }
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(row_ind));
    CUDA_CHECK(hipFree(row_ind_ptr));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    CUDA_CHECK(hipFree(mask));
    CUDA_CHECK(hipFree(m));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  WeakCCInputs<Index_> params;
  hipStream_t stream;
  Index_ *row_ind, *row_ind_ptr, *result, *verify;
  bool *mask, *m;
};

using WeakCCTestI = WeakCCTest<int>;
TEST_P(WeakCCTestI, Result) { Run(); }

using WeakCCTestL = WeakCCTest<int64_t>;
TEST_P(WeakCCTestL, Result) { Run(); }

// Hand-designed corner cases for weakcc
const std::vector<WeakCCInputs<int>> weakcc_inputs_32 = {
  {6,
   {1, 0, 1, 1, 1, 0},
   {{{0, 2, 5, 7}, {0, 1, 0, 1, 4, 2, 5}},
    {{0, 2, 5, 7}, {3, 4, 1, 3, 4, 2, 5}}},
   {{1, 1, 3, 4, 5, 3}, {1, 4, 3, 4, 4, 3}}},
  {6,
   {1, 0, 1, 0, 1, 0},
   {{{0, 5, 8}, {0, 1, 2, 3, 4, 0, 1, 4}},
    {{0, 5, 8}, {0, 2, 3, 4, 5, 0, 2, 3}},
    {{0, 5, 8}, {0, 1, 2, 4, 5, 2, 4, 5}}},
   {{1, 1, 1, 1, 1, MAX32}, {1, MAX32, 1, 1, 1, 1}, {1, 1, 1, MAX32, 1, 1}}},
  {6,
   {1, 1, 1, 0, 1, 1},
   {{{0, 3, 6}, {0, 1, 2, 0, 1, 3}},
    {{0, 3, 6}, {0, 2, 4, 1, 3, 5}},
    {{0, 3, 6}, {2, 4, 5, 3, 4, 5}}},
   {{1, 1, 1, 1, 5, 6}, {1, 2, 1, 2, 1, 6}, {1, 2, 3, 3, 3, 3}}},
  {8,
   {1, 1, 1, 1, 0, 0, 1, 1},
   {{{0, 2, 5}, {0, 1, 0, 1, 2}},
    {{0, 3, 6}, {1, 2, 3, 2, 3, 4}},
    {{0, 2, 4}, {3, 4, 5, 6}},
    {{0, 2, 5}, {5, 6, 7, 6, 7}}},
   {{1, 1, 1, 4, MAX32, MAX32, 7, 8},
    {1, 2, 2, 2, 2, MAX32, 7, 8},
    {1, 2, 3, 4, 4, 7, 7, 8},
    {1, 2, 3, 4, MAX32, 7, 7, 7}}}};
const std::vector<WeakCCInputs<int64_t>> weakcc_inputs_64 = {
  {6,
   {1, 0, 1, 1, 1, 0},
   {{{0, 2, 5, 7}, {0, 1, 0, 1, 4, 2, 5}},
    {{0, 2, 5, 7}, {3, 4, 1, 3, 4, 2, 5}}},
   {{1, 1, 3, 4, 5, 3}, {1, 4, 3, 4, 4, 3}}},
  {6,
   {1, 0, 1, 0, 1, 0},
   {{{0, 5, 8}, {0, 1, 2, 3, 4, 0, 1, 4}},
    {{0, 5, 8}, {0, 2, 3, 4, 5, 0, 2, 3}},
    {{0, 5, 8}, {0, 1, 2, 4, 5, 2, 4, 5}}},
   {{1, 1, 1, 1, 1, MAX64}, {1, MAX64, 1, 1, 1, 1}, {1, 1, 1, MAX64, 1, 1}}},
  {6,
   {1, 1, 1, 0, 1, 1},
   {{{0, 3, 6}, {0, 1, 2, 0, 1, 3}},
    {{0, 3, 6}, {0, 2, 4, 1, 3, 5}},
    {{0, 3, 6}, {2, 4, 5, 3, 4, 5}}},
   {{1, 1, 1, 1, 5, 6}, {1, 2, 1, 2, 1, 6}, {1, 2, 3, 3, 3, 3}}},
  {8,
   {1, 1, 1, 1, 0, 0, 1, 1},
   {{{0, 2, 5}, {0, 1, 0, 1, 2}},
    {{0, 3, 6}, {1, 2, 3, 2, 3, 4}},
    {{0, 2, 4}, {3, 4, 5, 6}},
    {{0, 2, 5}, {5, 6, 7, 6, 7}}},
   {{1, 1, 1, 4, MAX64, MAX64, 7, 8},
    {1, 2, 2, 2, 2, MAX64, 7, 8},
    {1, 2, 3, 4, 4, 7, 7, 8},
    {1, 2, 3, 4, MAX64, 7, 7, 7}}}};

/**************************** Test instantiation ****************************/

INSTANTIATE_TEST_CASE_P(CSRTests, CSRtoCOOTestI,
                        ::testing::ValuesIn(csrtocoo_inputs_32));
INSTANTIATE_TEST_CASE_P(CSRTests, CSRtoCOOTestL,
                        ::testing::ValuesIn(csrtocoo_inputs_64));

INSTANTIATE_TEST_CASE_P(CSRTests, CSRRowNormalizeTestF,
                        ::testing::ValuesIn(csrnormalize_inputs_f));
INSTANTIATE_TEST_CASE_P(CSRTests, CSRRowNormalizeTestD,
                        ::testing::ValuesIn(csrnormalize_inputs_d));

INSTANTIATE_TEST_CASE_P(CSRTests, CSRSumTestF,
                        ::testing::ValuesIn(csrsum_inputs_f));
INSTANTIATE_TEST_CASE_P(CSRTests, CSRSumTestD,
                        ::testing::ValuesIn(csrsum_inputs_d));

INSTANTIATE_TEST_CASE_P(CSRTests, CSRRowOpTestF,
                        ::testing::ValuesIn(csrrowop_inputs_f));
INSTANTIATE_TEST_CASE_P(CSRTests, CSRRowOpTestD,
                        ::testing::ValuesIn(csrrowop_inputs_d));

INSTANTIATE_TEST_CASE_P(CSRTests, CSRAdjGraphTestI,
                        ::testing::ValuesIn(csradjgraph_inputs_i));
INSTANTIATE_TEST_CASE_P(CSRTests, CSRAdjGraphTestL,
                        ::testing::ValuesIn(csradjgraph_inputs_l));

INSTANTIATE_TEST_CASE_P(CSRTests, WeakCCTestI,
                        ::testing::ValuesIn(weakcc_inputs_32));
INSTANTIATE_TEST_CASE_P(CSRTests, WeakCCTestL,
                        ::testing::ValuesIn(weakcc_inputs_64));

}  // namespace Sparse
}  // namespace MLCommon
