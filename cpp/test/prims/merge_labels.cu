#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <label/merge_labels.cuh>

#include <raft/cudart_utils.h>
#include <thrust/device_ptr.h>
#include <cuml/common/cuml_allocator.hpp>
#include <rmm/device_vector.hpp>
#include "test_utils.h"

#include <vector>

namespace MLCommon {
namespace Label {

template <typename Index_>
struct MergeLabelsInputs {
  Index_ N;
  std::vector<Index_> labels_a;
  std::vector<Index_> labels_b;
  std::vector<uint8_t> mask;  // to avoid std::vector<bool> optimization
  std::vector<Index_> expected;
};

template <typename Index_>
class MergeLabelsTest
  : public ::testing::TestWithParam<MergeLabelsInputs<Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MergeLabelsInputs<Index_>>::GetParam();

    CUDA_CHECK(hipStreamCreate(&stream));
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);

    labels_a.resize(params.N);
    labels_b.resize(params.N);
    expected.resize(params.N);
    R.resize(params.N);
    mask.resize(params.N);
    m.resize(1);
  }

  void Run() {
    params = ::testing::TestWithParam<MergeLabelsInputs<Index_>>::GetParam();

    raft::update_device(thrust::raw_pointer_cast(labels_a.data()),
                        params.labels_a.data(), params.N, stream);
    raft::update_device(thrust::raw_pointer_cast(labels_b.data()),
                        params.labels_b.data(), params.N, stream);
    raft::update_device(thrust::raw_pointer_cast(expected.data()),
                        params.expected.data(), params.N, stream);
    raft::update_device(thrust::raw_pointer_cast(mask.data()),
                        reinterpret_cast<bool *>(params.mask.data()), params.N,
                        stream);

    merge_labels(thrust::raw_pointer_cast(labels_a.data()),
                 thrust::raw_pointer_cast(labels_b.data()),
                 thrust::raw_pointer_cast(mask.data()),
                 thrust::raw_pointer_cast(R.data()),
                 thrust::raw_pointer_cast(m.data()), params.N, stream);

    hipStreamSynchronize(stream);
    ASSERT_TRUE(
      raft::devArrMatch<Index_>(thrust::raw_pointer_cast(expected.data()),
                                thrust::raw_pointer_cast(labels_a.data()),
                                params.N, raft::Compare<Index_>()));
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }

 protected:
  MergeLabelsInputs<Index_> params;
  hipStream_t stream;
  rmm::device_vector<Index_> labels_a, labels_b, expected, R;
  rmm::device_vector<bool> mask, m;
};

using MergeLabelsTestI = MergeLabelsTest<int>;
TEST_P(MergeLabelsTestI, Result) { Run(); }

using MergeLabelsTestL = MergeLabelsTest<int64_t>;
TEST_P(MergeLabelsTestL, Result) { Run(); }

constexpr int MAX32 = std::numeric_limits<int>::max();
constexpr int64_t MAX64 = std::numeric_limits<int64_t>::max();

const std::vector<MergeLabelsInputs<int>> merge_inputs_32 = {
  {4, {1, 1, 3, MAX32}, {1, 3, 3, 1}, {1, 0, 1, 0}, {1, 1, 3, 1}},
  {5, {1, 2, 2, 2, 1}, {4, 2, 4, 4, 4}, {1, 1, 1, 1, 1}, {1, 1, 1, 1, 1}},
  {6,
   {1, 2, 1, 4, 5, MAX32},
   {1, 2, MAX32, 4, 5, 4},
   {1, 1, 0, 1, 1, 0},
   {1, 2, 1, 4, 5, 4}},
  {6,
   {1, 2, 2, 2, 2, 6},
   {1, 1, 1, 5, 5, 5},
   {1, 1, 1, 1, 1, 1},
   {1, 1, 1, 1, 1, 1}},
  {8,
   {1, 1, 3, 3, MAX32, 1, 3, MAX32},
   {1, 2, 3, 2, MAX32, 2, 2, 2},
   {1, 1, 1, 1, 0, 1, 1, 0},
   {1, 1, 1, 1, MAX32, 1, 1, 1}},
  {8,
   {1, 1, 1, 4, 4, 7, 7, 8},
   {1, 2, 2, 2, 2, 7, 7, 7},
   {1, 1, 1, 1, 0, 0, 1, 1},
   {1, 1, 1, 1, 1, 7, 7, 7}},
};

const std::vector<MergeLabelsInputs<int64_t>> merge_inputs_64 = {
  {4, {1, 1, 3, MAX64}, {1, 3, 3, 1}, {1, 0, 1, 0}, {1, 1, 3, 1}},
  {5, {1, 2, 2, 2, 1}, {4, 2, 4, 4, 4}, {1, 1, 1, 1, 1}, {1, 1, 1, 1, 1}},
  {6,
   {1, 2, 1, 4, 5, MAX64},
   {1, 2, MAX64, 4, 5, 4},
   {1, 1, 0, 1, 1, 0},
   {1, 2, 1, 4, 5, 4}},
  {6,
   {1, 2, 2, 2, 2, 6},
   {1, 1, 1, 5, 5, 5},
   {1, 1, 1, 1, 1, 1},
   {1, 1, 1, 1, 1, 1}},
  {8,
   {1, 1, 3, 3, MAX64, 1, 3, MAX64},
   {1, 2, 3, 2, MAX64, 2, 2, 2},
   {1, 1, 1, 1, 0, 1, 1, 0},
   {1, 1, 1, 1, MAX64, 1, 1, 1}},
  {8,
   {1, 1, 1, 4, 4, 7, 7, 8},
   {1, 2, 2, 2, 2, 7, 7, 7},
   {1, 1, 1, 1, 0, 0, 1, 1},
   {1, 1, 1, 1, 1, 7, 7, 7}},
};

INSTANTIATE_TEST_CASE_P(MergeLabelsTests, MergeLabelsTestI,
                        ::testing::ValuesIn(merge_inputs_32));
INSTANTIATE_TEST_CASE_P(MergeLabelsTests, MergeLabelsTestL,
                        ::testing::ValuesIn(merge_inputs_64));

}  // namespace Label
}  // namespace MLCommon
