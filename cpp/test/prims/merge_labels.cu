#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <label/merge_labels.cuh>

#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include "test_utils.h"

// #include <iostream>
// #include <limits>
#include <vector>

namespace MLCommon {
namespace Label {

template <typename Index_>
struct MergeLabelsInputs {
  Index_ N;
  std::vector<Index_> labelsA;
  std::vector<Index_> labelsB;
  std::vector<uint8_t> mask;  // to avoid std::vector<bool> optimization
  std::vector<Index_> expected;
};

template <typename Index_>
class MergeLabelsTest
  : public ::testing::TestWithParam<MergeLabelsInputs<Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MergeLabelsInputs<Index_>>::GetParam();

    CUDA_CHECK(hipStreamCreate(&stream));
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);

    raft::allocate(labelsA, params.N);
    raft::allocate(labelsB, params.N);
    raft::allocate(expected, params.N);
    raft::allocate(R, params.N);
    raft::allocate(mask, params.N);
    raft::allocate(m, 1);
  }

  void Run() {
    params = ::testing::TestWithParam<MergeLabelsInputs<Index_>>::GetParam();

    raft::update_device(labelsA, params.labelsA.data(), params.N, stream);
    raft::update_device(labelsB, params.labelsB.data(), params.N, stream);
    raft::update_device(expected, params.expected.data(), params.N, stream);
    raft::update_device(mask, reinterpret_cast<bool *>(params.mask.data()),
                        params.N, stream);

    merge_labels(labelsA, labelsB, mask, R, m, params.N, stream);

    hipStreamSynchronize(stream);
    ASSERT_TRUE(raft::devArrMatch<Index_>(expected, labelsA, params.N,
                                          raft::Compare<Index_>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(labelsA));
    CUDA_CHECK(hipFree(labelsB));
    CUDA_CHECK(hipFree(expected));
    CUDA_CHECK(hipFree(R));
    CUDA_CHECK(hipFree(mask));
    CUDA_CHECK(hipFree(m));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  MergeLabelsInputs<Index_> params;
  hipStream_t stream;
  Index_ *labelsA, *labelsB, *expected, *R;
  bool *mask, *m;
};

using MergeLabelsTestI = MergeLabelsTest<int>;
TEST_P(MergeLabelsTestI, Result) { Run(); }

using MergeLabelsTestL = MergeLabelsTest<int64_t>;
TEST_P(MergeLabelsTestL, Result) { Run(); }

constexpr int MAX32 = std::numeric_limits<int>::max();
constexpr int64_t MAX64 = std::numeric_limits<int64_t>::max();

const std::vector<MergeLabelsInputs<int>> merge_inputs_32 = {
  {4, {1, 1, 3, MAX32}, {1, 3, 3, 1}, {1, 0, 1, 0}, {1, 1, 3, 1}},
  {5, {1, 2, 2, 2, 1}, {4, 2, 4, 4, 4}, {1, 1, 1, 1, 1}, {1, 1, 1, 1, 1}},
  {6,
   {1, 2, 1, 4, 5, MAX32},
   {1, 2, MAX32, 4, 5, 4},
   {1, 1, 0, 1, 1, 0},
   {1, 2, 1, 4, 5, 4}},
  {6,
   {1, 2, 2, 2, 2, 6},
   {1, 1, 1, 5, 5, 5},
   {1, 1, 1, 1, 1, 1},
   {1, 1, 1, 1, 1, 1}},
  {8,
   {1, 1, 3, 3, MAX32, 1, 3, MAX32},
   {1, 2, 3, 2, MAX32, 2, 2, 2},
   {1, 1, 1, 1, 0, 1, 1, 0},
   {1, 1, 1, 1, MAX32, 1, 1, 1}},
  {8,
   {1, 1, 1, 4, 4, 7, 7, 8},
   {1, 2, 2, 2, 2, 7, 7, 7},
   {1, 1, 1, 1, 0, 0, 1, 1},
   {1, 1, 1, 1, 1, 7, 7, 7}},
};

const std::vector<MergeLabelsInputs<int64_t>> merge_inputs_64 = {
  {4, {1, 1, 3, MAX64}, {1, 3, 3, 1}, {1, 0, 1, 0}, {1, 1, 3, 1}},
  {5, {1, 2, 2, 2, 1}, {4, 2, 4, 4, 4}, {1, 1, 1, 1, 1}, {1, 1, 1, 1, 1}},
  {6,
   {1, 2, 1, 4, 5, MAX64},
   {1, 2, MAX64, 4, 5, 4},
   {1, 1, 0, 1, 1, 0},
   {1, 2, 1, 4, 5, 4}},
  {6,
   {1, 2, 2, 2, 2, 6},
   {1, 1, 1, 5, 5, 5},
   {1, 1, 1, 1, 1, 1},
   {1, 1, 1, 1, 1, 1}},
  {8,
   {1, 1, 3, 3, MAX64, 1, 3, MAX64},
   {1, 2, 3, 2, MAX64, 2, 2, 2},
   {1, 1, 1, 1, 0, 1, 1, 0},
   {1, 1, 1, 1, MAX64, 1, 1, 1}},
  {8,
   {1, 1, 1, 4, 4, 7, 7, 8},
   {1, 2, 2, 2, 2, 7, 7, 7},
   {1, 1, 1, 1, 0, 0, 1, 1},
   {1, 1, 1, 1, 1, 7, 7, 7}},
};

INSTANTIATE_TEST_CASE_P(MergeLabelsTests, MergeLabelsTestI,
                        ::testing::ValuesIn(merge_inputs_32));
INSTANTIATE_TEST_CASE_P(MergeLabelsTests, MergeLabelsTestL,
                        ::testing::ValuesIn(merge_inputs_64));

}  // namespace Label
}  // namespace MLCommon
